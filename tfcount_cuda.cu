#include "hip/hip_runtime.h"
#undef _GLIBCXX_USE_INT128
#undef _GLIBCXX_ATOMIC_BUILTINS

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <thrust/count.h>
#include <thrust/reduce.h>
#include <thrust/sequence.h>
#include <thrust/transform.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>
#include <thrust/partition.h>

extern "C" {
#include <bcutils/bcutils.h>
}

#include "tfcount_cuda.h"


// Sequence handling
#include <zlib.h>
#include <bcutils/kseq.h>
KSEQ_INIT(gzFile, gzread)

#define MAX_THREADS_PER_BLOCK 1024
#define SCORE_THREADS_PER_BLOCK 448
#define TALLY_THREADS_PER_BLOCK 768
#define MAX_BLOCKS_PER_GRID 65535

#define PADDED_RVD_WIDTH 32

#define cudaSafeCall(call){   \
  hipError_t err = call;       \
  if(hipSuccess != err){     \
    fprintf(stderr, "%s(%i) : %s.\n", __FILE__, __LINE__, hipGetErrorString(err));   \
    exit(EXIT_FAILURE);       \
}}

__device__ double ScoringMatrixVal(double *scoring_matrix, size_t pitch, unsigned int row, unsigned int column);
double *ScoringMatrixRow(double *scoring_matrix, size_t pitch, unsigned int row);

__global__ void ScoreBindingSites(char *input_sequence, unsigned long is_length, unsigned int *rvd_sequence, unsigned int rs_len, double cutoff, int c_upstream, unsigned int rvd_num, double *scoring_matrix, size_t sm_pitch, unsigned char *results) {

  int block_seq_index = SCORE_THREADS_PER_BLOCK * (blockIdx.y * gridDim.x + blockIdx.x);
  int thread_id = (blockDim.x * threadIdx.y) + threadIdx.x;
  int seq_index = block_seq_index + thread_id + 1;

  if (seq_index < 1 || seq_index >= is_length || seq_index + rs_len >= is_length - 1) return;

  char first = input_sequence[seq_index - 1];
  char last  = input_sequence[seq_index + rs_len];

  int first_t = c_upstream != 1 && (first == 'T' || first == 't');
  int first_c = c_upstream != 0 && (first == 'C' || first == 'c');
  int last_a  = c_upstream != 1 && (last == 'A' || last == 'a');
  int last_g  = c_upstream != 0 && (last == 'G' || last == 'g');


  if (first_c || first_t || last_g || last_a) {

    double thread_result_t = 0;
    double thread_result_a = 0;

    for (int i = 0; i < rs_len; i++) {

      int sm_col_t = 4;

      char base = input_sequence[seq_index + i];

      if (base == 'A' || base == 'a')
        sm_col_t = 0;
      if (base == 'C' || base == 'c')
        sm_col_t = 1;
      if (base == 'G' || base == 'g')
        sm_col_t = 2;
      if (base == 'T' || base == 't')
        sm_col_t = 3;

      int rvd_index_t = i;
      int rvd_index_a = rs_len - i - 1;

      thread_result_t += ScoringMatrixVal(scoring_matrix, sm_pitch, rvd_sequence[rvd_index_t], sm_col_t);

      int sm_col_a = (sm_col_t == 4 ? 4 : 3 - sm_col_t);

      thread_result_a += ScoringMatrixVal(scoring_matrix, sm_pitch, rvd_sequence[rvd_index_a], sm_col_a);

    }

    if (first_c || first_t)
      results[seq_index] |= (thread_result_t < cutoff ? 1UL : 0UL) << ((2 * rvd_num) + (first_c * 4));

    if (last_g || last_a)
      results[seq_index] |= (thread_result_a < cutoff ? 1UL : 0UL) << ((2 * rvd_num + 1) + (last_g * 4));

  }

}

__global__ void TallyResults(unsigned char *prelim_results, unsigned int pr_length, unsigned int rs_len, int c_upstream, unsigned int u_shift, unsigned int d_shift, unsigned int spacer_range_start, unsigned int spacer_range_end, unsigned int *second_results) {
    
  short thread_result = 0;
  
  int block_seq_index = TALLY_THREADS_PER_BLOCK * (blockIdx.y * gridDim.x + blockIdx.x);
  int seq_index = block_seq_index + (blockDim.x * threadIdx.y) + threadIdx.x;
  
  if (seq_index < 0 || seq_index >= pr_length) return;

  int first_t = (prelim_results[seq_index] & (1UL << u_shift)) > 0;
  int first_c = (prelim_results[seq_index] & (1UL << (u_shift + 4))) > 0;

  if (!((c_upstream != 0 && first_c) || (c_upstream != 1 && first_t))) return;
  
  for (int i = spacer_range_start; i <= spacer_range_end; i++) {
    
    if (seq_index + rs_len + i >= pr_length) continue;
    
    thread_result += ((prelim_results[seq_index + rs_len + i] & (1UL << (d_shift + (first_c * 4))) ) > 0);

  }
  
  second_results[seq_index] = thread_result;
  
}

__device__ double ScoringMatrixVal(double *scoring_matrix, size_t pitch, unsigned int row, unsigned int column) {
  
  return *((double*)((char*) scoring_matrix + row * pitch) + column);
  
}

double *ScoringMatrixRow(double *scoring_matrix, size_t pitch, unsigned int row) {
  return (double*)((char*) scoring_matrix + row * pitch);
}

struct first_t_or_c
{
  __host__ __device__ bool operator()(const unsigned char &x) {
      return (x & 1) || (x & (1 << 4));
  }
};



struct last_a_or_g
{
  __host__ __device__ bool operator()(const unsigned char &x) {
      return (x & (1 << 1)) || (x & (1 << 5));
  }
};

struct index_to_keep
{
  const int offset;

  index_to_keep(int _offset) : offset(_offset) {}

  __host__ __device__ int operator()(const unsigned char &x, const int &y) {
      if ((x & 1) || (x & (1 << 4)) || (x & (1 << 1)) || (x & (1 << 5))) {
        return y + offset;
      } else {
        return -1;
      }
  }
};

struct index_to_keep_paired
{
  const int offset;

  index_to_keep_paired(int _offset) : offset(_offset) {}

  __host__ __device__ int operator()(const unsigned char &x, const int &y) {
      if ((x & 1) || (x & (1 << 4)) || (x & (1 << 2)) || (x & (1 << 6))) {
        return y + offset;
      } else {
        return -1;
      }
  }
};

struct valid_index
{
  __host__ __device__ bool operator()(const int &x) {
       return x > -1;
  }
};

void RunCountBindingSites(char *seq_filename, FILE *log_file, unsigned int *rvd_seqs, unsigned int *rvd_lengths, double *cutoffs, unsigned int num_rvd_seqs, int c_upstream, double **scoring_matrix, unsigned int scoring_matrix_length, unsigned int *results) {
  
  unsigned int *d_rvd_seqs;
  double *d_scoring_matrix;
  size_t sm_pitch;
  
  cudaSafeCall( hipMalloc(&d_rvd_seqs, PADDED_RVD_WIDTH * num_rvd_seqs * sizeof(unsigned int)));
  cudaSafeCall( hipMemcpy(d_rvd_seqs, rvd_seqs, PADDED_RVD_WIDTH * num_rvd_seqs * sizeof(unsigned int), hipMemcpyHostToDevice) );
  
  cudaSafeCall( hipMallocPitch(&d_scoring_matrix, &sm_pitch, 5 * sizeof(double), scoring_matrix_length * sizeof(double)) );
  
  for (unsigned int i = 0; i < scoring_matrix_length; i++) {
    cudaSafeCall( hipMemcpy(ScoringMatrixRow(d_scoring_matrix, sm_pitch, i), scoring_matrix[i], sizeof(double) * 5, hipMemcpyHostToDevice) );
  }
  
  dim3 score_threadsPerBlock(32, 14);
  
  gzFile seqfile = gzopen(seq_filename, "r");

  kseq_t *seq = kseq_init(seqfile);
  int result;

  while ((result = kseq_read(seq)) >= 0) {

    unsigned char *d_prelim_results;
    char *d_reference_sequence;

    char *reference_sequence = seq->seq.s;
    unsigned long reference_sequence_length = ((seq->seq.l + 31) / 32 ) * 32;
    
    for (unsigned long i = seq->seq.l; i < reference_sequence_length - 1; i++) {
      reference_sequence[i] = 'X';
    }
    
    reference_sequence[reference_sequence_length- 1] = '\0';

    logger(log_file, "Scanning %s for off-target sites (length %ld)", seq->name.s, seq->seq.l);

    cudaSafeCall( hipMalloc(&d_reference_sequence, reference_sequence_length * sizeof(char)) );
    cudaSafeCall( hipMemcpy(d_reference_sequence, reference_sequence, reference_sequence_length * sizeof(char), hipMemcpyHostToDevice) );

    cudaSafeCall( hipMalloc(&d_prelim_results, reference_sequence_length * sizeof(unsigned char)) );
    
    thrust::device_ptr<unsigned char> prelim_results_start(d_prelim_results);
    thrust::device_ptr<unsigned char> prelim_results_end(d_prelim_results + reference_sequence_length);

    int score_blocks_needed = (reference_sequence_length + SCORE_THREADS_PER_BLOCK - 1) / SCORE_THREADS_PER_BLOCK;

    int score_block_x = (score_blocks_needed >= MAX_BLOCKS_PER_GRID ? MAX_BLOCKS_PER_GRID : score_blocks_needed);
    int score_block_y = (score_blocks_needed + (MAX_BLOCKS_PER_GRID - 1)) / MAX_BLOCKS_PER_GRID;

    dim3 score_blocksPerGrid(score_block_x, score_block_y);
    
    for (int i = 0; i < num_rvd_seqs; i++) {
      
      cudaSafeCall( hipMemset(d_prelim_results, '\0', reference_sequence_length * sizeof(unsigned char)) );
      
      ScoreBindingSites <<<score_blocksPerGrid, score_threadsPerBlock>>>(d_reference_sequence, reference_sequence_length, d_rvd_seqs + i * PADDED_RVD_WIDTH, rvd_lengths[i], cutoffs[i], c_upstream, 0, d_scoring_matrix, sm_pitch, d_prelim_results);
      cudaSafeCall( hipGetLastError() );
      
      results[i] += thrust::count_if(prelim_results_start, prelim_results_end, first_t_or_c());
      results[i] += thrust::count_if(prelim_results_start, prelim_results_end, last_a_or_g());

      
    }

    cudaSafeCall( hipFree(d_prelim_results) );
    cudaSafeCall( hipFree(d_reference_sequence) );
    
  }

  kseq_destroy(seq);
  gzclose(seqfile);
  
  cudaSafeCall( hipFree(d_rvd_seqs) );
  cudaSafeCall( hipFree(d_scoring_matrix) );
  
}

void RunPairedCountBindingSites(char *seq_filename, FILE *log_file, unsigned int *spacer_sizes, unsigned int *rvd_pairs, unsigned int *rvd_lengths, double *cutoffs, unsigned int num_rvd_pairs, int c_upstream, double **scoring_matrix, unsigned int scoring_matrix_length, unsigned int *results) {
  
  unsigned int *d_rvd_pairs;
  double *d_scoring_matrix;
  size_t sm_pitch;
  
  cudaSafeCall( hipMalloc(&d_rvd_pairs, 2 * PADDED_RVD_WIDTH * num_rvd_pairs * sizeof(unsigned int)));
  cudaSafeCall( hipMemcpy(d_rvd_pairs, rvd_pairs, 2 * PADDED_RVD_WIDTH * num_rvd_pairs * sizeof(unsigned int), hipMemcpyHostToDevice) );
  
  cudaSafeCall( hipMallocPitch(&d_scoring_matrix, &sm_pitch, 5 * sizeof(double), scoring_matrix_length * sizeof(double)) );
  
  for (unsigned int i = 0; i < scoring_matrix_length; i++) {
    cudaSafeCall( hipMemcpy(ScoringMatrixRow(d_scoring_matrix, sm_pitch, i), scoring_matrix[i], sizeof(double) * 5, hipMemcpyHostToDevice) );
  }
  
  dim3 score_threadsPerBlock(32, 14);
  dim3 tally_threadsPerBlock(32, 24);
  
  gzFile seqfile = gzopen(seq_filename, "r");

  kseq_t *seq = kseq_init(seqfile);
  int result;

  while ((result = kseq_read(seq)) >= 0) {

    unsigned char *d_prelim_results;
    unsigned int *d_second_results;
    char *d_reference_sequence;

    char *reference_sequence = seq->seq.s;
    unsigned long reference_sequence_length = ((seq->seq.l + 31) / 32 ) * 32;
    
    for (unsigned long i = seq->seq.l; i < reference_sequence_length - 1; i++) {
      reference_sequence[i] = 'X';
    }
    
    reference_sequence[reference_sequence_length- 1] = '\0';

    logger(log_file, "Scanning %s for off-target sites (length %ld)", seq->name.s, seq->seq.l);

    cudaSafeCall( hipMalloc(&d_reference_sequence, reference_sequence_length * sizeof(char)) );
    cudaSafeCall( hipMemcpy(d_reference_sequence, reference_sequence, reference_sequence_length * sizeof(char), hipMemcpyHostToDevice) );

    cudaSafeCall( hipMalloc(&d_prelim_results, reference_sequence_length * sizeof(unsigned char)) );
    cudaSafeCall( hipMalloc(&d_second_results, reference_sequence_length * sizeof(unsigned int)) );
    
    thrust::device_ptr<unsigned int> second_results_start(d_second_results);
    thrust::device_ptr<unsigned int> second_results_end(d_second_results + reference_sequence_length);

    int score_blocks_needed = (reference_sequence_length + SCORE_THREADS_PER_BLOCK - 1) / SCORE_THREADS_PER_BLOCK;

    int score_block_x = (score_blocks_needed >= MAX_BLOCKS_PER_GRID ? MAX_BLOCKS_PER_GRID : score_blocks_needed);
    int score_block_y = (score_blocks_needed + (MAX_BLOCKS_PER_GRID - 1)) / MAX_BLOCKS_PER_GRID;

    dim3 score_blocksPerGrid(score_block_x, score_block_y);

    int tally_blocks_needed = (reference_sequence_length + TALLY_THREADS_PER_BLOCK - 1) / TALLY_THREADS_PER_BLOCK;

    int tally_block_x = (tally_blocks_needed >= MAX_BLOCKS_PER_GRID ? MAX_BLOCKS_PER_GRID : tally_blocks_needed);
    int tally_block_y = (tally_blocks_needed + (MAX_BLOCKS_PER_GRID - 1)) / MAX_BLOCKS_PER_GRID;

    dim3 tally_blocksPerGrid(tally_block_x, tally_block_y);
    
    for (int i = 0; i < num_rvd_pairs; i++) {
      
      unsigned int *pair_final_results = results + (4 * i);
      unsigned int pair_temp_results[4];
      
      cudaSafeCall( hipMemset(d_prelim_results, '\0', reference_sequence_length * sizeof(unsigned char)) );
      cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
      
      int first_index = 2 * i;
      int second_index = first_index + 1;

      ScoreBindingSites <<<score_blocksPerGrid, score_threadsPerBlock>>>(d_reference_sequence, reference_sequence_length, d_rvd_pairs + first_index * PADDED_RVD_WIDTH, rvd_lengths[first_index], cutoffs[first_index], c_upstream, 0, d_scoring_matrix, sm_pitch, d_prelim_results);
      cudaSafeCall( hipGetLastError() );

      ScoreBindingSites <<<score_blocksPerGrid, score_threadsPerBlock>>>(d_reference_sequence, reference_sequence_length, d_rvd_pairs + second_index * PADDED_RVD_WIDTH, rvd_lengths[second_index], cutoffs[second_index], c_upstream, 1, d_scoring_matrix, sm_pitch, d_prelim_results);
      cudaSafeCall( hipGetLastError() );

      TallyResults<<<tally_blocksPerGrid, tally_threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_lengths[first_index], c_upstream, 0, 1, spacer_sizes[0], spacer_sizes[1], d_second_results);
      cudaSafeCall( hipGetLastError() );
      
      pair_temp_results[0] = thrust::reduce(second_results_start, second_results_end);
      cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
      
      TallyResults<<<tally_blocksPerGrid, tally_threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_lengths[first_index], c_upstream, 0, 3, spacer_sizes[0], spacer_sizes[1], d_second_results);
      cudaSafeCall( hipGetLastError() );
      
      pair_temp_results[1] = thrust::reduce(second_results_start, second_results_end);
      cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
      
      TallyResults<<<tally_blocksPerGrid, tally_threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_lengths[second_index], c_upstream, 2, 1, spacer_sizes[0], spacer_sizes[1], d_second_results);
      cudaSafeCall( hipGetLastError() );
      
      pair_temp_results[2] = thrust::reduce(second_results_start, second_results_end);
      cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
      
      TallyResults<<<tally_blocksPerGrid, tally_threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_lengths[second_index], c_upstream, 2, 3, spacer_sizes[0], spacer_sizes[1], d_second_results);
      cudaSafeCall( hipGetLastError() );
      
      pair_temp_results[3] = thrust::reduce(second_results_start, second_results_end);
      
      pair_final_results[0] += pair_temp_results[0];
      pair_final_results[1] += pair_temp_results[1];
      pair_final_results[2] += pair_temp_results[2];
      pair_final_results[3] += pair_temp_results[3];
      
    }

    cudaSafeCall( hipFree(d_prelim_results) );
    cudaSafeCall( hipFree(d_second_results) );
    cudaSafeCall( hipFree(d_reference_sequence) );
    
  }

  kseq_destroy(seq);
  gzclose(seqfile);
  
  cudaSafeCall( hipFree(d_rvd_pairs) );
  cudaSafeCall( hipFree(d_scoring_matrix) );
  
}

void RunPairedFindBindingSitesKeepScores_init(unsigned int **d_rvd_pair_p, double **d_scoring_matrix_p, size_t *sm_pitch_p, unsigned char **d_prelim_results_p, int **d_prelim_results_indexes_p, char **d_reference_sequence_p, unsigned char **prelim_results_p, int **prelim_results_indexes_p, unsigned long *reference_window_size_p, int *score_block_x_p, int *score_block_y_p, unsigned int **rvd_pair, double **scoring_matrix, unsigned int *rvd_lengths, unsigned int scoring_matrix_length) {
  
  unsigned int *d_rvd_pair;
  double *d_scoring_matrix;
  size_t sm_pitch;
  unsigned char *d_prelim_results;
  int *d_prelim_results_indexes;
  unsigned char *prelim_results;
  int *prelim_results_indexes;
  char *d_reference_sequence;
  // must be divisible by 32, pref power of 2
  unsigned long reference_window_size = 134217728;// 2^27
  int score_block_x;
  int score_block_y;
  
  cudaSafeCall( hipMalloc(&d_rvd_pair, 2 * PADDED_RVD_WIDTH * sizeof(unsigned int)));
  cudaSafeCall( hipMemcpy(d_rvd_pair, rvd_pair[0], rvd_lengths[0] * sizeof(unsigned int), hipMemcpyHostToDevice) );
  cudaSafeCall( hipMemcpy(d_rvd_pair + PADDED_RVD_WIDTH, rvd_pair[1], rvd_lengths[1] * sizeof(unsigned int), hipMemcpyHostToDevice) );
  
  cudaSafeCall( hipMallocPitch(&d_scoring_matrix, &sm_pitch, 5 * sizeof(double), scoring_matrix_length * sizeof(double)) );
  
  for (unsigned int i = 0; i < scoring_matrix_length; i++) {
    cudaSafeCall( hipMemcpy(ScoringMatrixRow(d_scoring_matrix, sm_pitch, i), scoring_matrix[i], sizeof(double) * 5, hipMemcpyHostToDevice) );
  }
  
  cudaSafeCall( hipMalloc(&d_reference_sequence, reference_window_size * sizeof(char)) );
  cudaSafeCall( hipMalloc(&d_prelim_results, reference_window_size * sizeof(unsigned char)) );
  cudaSafeCall( hipMalloc(&d_prelim_results_indexes, reference_window_size * sizeof(int)) );
  
  cudaSafeCall( hipHostMalloc(&prelim_results, 400000000 * sizeof(unsigned char)) );
  cudaSafeCall( hipHostMalloc(&prelim_results_indexes, 400000000 * sizeof(int)) );

  int score_blocks_needed = (reference_window_size + SCORE_THREADS_PER_BLOCK - 1) / SCORE_THREADS_PER_BLOCK;

  score_block_x = (score_blocks_needed >= MAX_BLOCKS_PER_GRID ? MAX_BLOCKS_PER_GRID : score_blocks_needed);
  score_block_y = (score_blocks_needed + (MAX_BLOCKS_PER_GRID - 1)) / MAX_BLOCKS_PER_GRID;
  
  *d_rvd_pair_p = d_rvd_pair;
  *d_scoring_matrix_p = d_scoring_matrix;
  *sm_pitch_p = sm_pitch;
  *d_prelim_results_p = d_prelim_results;
  *d_prelim_results_indexes_p = d_prelim_results_indexes;
  *prelim_results_p = prelim_results;
  *prelim_results_indexes_p = prelim_results_indexes;
  *d_reference_sequence_p = d_reference_sequence;
  *reference_window_size_p = reference_window_size;
  *score_block_x_p = score_block_x;
  *score_block_y_p = score_block_y;
  
}

int RunPairedFindBindingSitesKeepScores(char *d_reference_sequence, unsigned int *d_rvd_pairs, double *d_scoring_matrix, size_t sm_pitch, unsigned char *d_prelim_results, int *d_prelim_results_indexes, unsigned char *prelim_results, int *prelim_results_indexes, unsigned long reference_window_size, int score_block_x, int score_block_y, unsigned int *rvd_lengths, char *ref_seq, unsigned long ref_seq_len, double *cutoffs, int c_upstream) {
  
  dim3 score_threadsPerBlock(32, 14);
  dim3 score_blocksPerGrid(score_block_x, score_block_y);
  
  thrust::device_ptr<int> prelim_results_indexes_start(d_prelim_results_indexes);
  thrust::device_ptr<int> prelim_results_indexes_end(d_prelim_results_indexes + reference_window_size);
  thrust::device_ptr<unsigned char> prelim_results_start(d_prelim_results);
  thrust::device_ptr<unsigned char> prelim_results_end(d_prelim_results + reference_window_size);

  int keepers_end_pos = 0;

  int max_rvd_len = (rvd_lengths[0] > rvd_lengths[1]) ? rvd_lengths[0] : rvd_lengths[1];
  int usable_tile_size = reference_window_size - (((max_rvd_len - 1) + 31) / 32 ) * 32;

  int iterations_needed = (ref_seq_len + (usable_tile_size - 1)) / usable_tile_size;
  
  memset(prelim_results, '\0', 400000000 * sizeof(unsigned char));
  
  for (int i = 0; i < iterations_needed; i++) {

    int copy_offset = usable_tile_size * i;

    int copy_num;

    if (ref_seq_len - copy_offset <= reference_window_size)
      copy_num = ref_seq_len - copy_offset;
    else
      copy_num = reference_window_size;

    cudaSafeCall( hipMemset(d_reference_sequence, 'X', reference_window_size * sizeof(unsigned char)) );
    cudaSafeCall( hipMemset(d_reference_sequence + reference_window_size - 1, '\0', 1 * sizeof(unsigned char)) );
    cudaSafeCall( hipMemcpy(d_reference_sequence, ref_seq + copy_offset, copy_num * sizeof(char), hipMemcpyHostToDevice) );

    cudaSafeCall( hipMemset(d_prelim_results, '\0', reference_window_size * sizeof(unsigned char)) );
    cudaSafeCall( hipMemset(d_prelim_results_indexes, '\0', reference_window_size * sizeof(unsigned int)) );
    
    ScoreBindingSites <<<score_blocksPerGrid, score_threadsPerBlock>>>(d_reference_sequence, reference_window_size, d_rvd_pairs + 0, rvd_lengths[0], cutoffs[0], c_upstream, 0, d_scoring_matrix, sm_pitch, d_prelim_results);
    cudaSafeCall( hipGetLastError() );

    ScoreBindingSites <<<score_blocksPerGrid, score_threadsPerBlock>>>(d_reference_sequence, reference_window_size, d_rvd_pairs + PADDED_RVD_WIDTH, rvd_lengths[1], cutoffs[1], c_upstream, 1, d_scoring_matrix, sm_pitch, d_prelim_results);
    cudaSafeCall( hipGetLastError() );

    cudaSafeCall( hipMemcpy(prelim_results + copy_offset, d_prelim_results, copy_num * sizeof(unsigned char), hipMemcpyDeviceToHost) );
    cudaSafeCall( hipGetLastError() );

    thrust::sequence(prelim_results_indexes_start, prelim_results_indexes_end);
    thrust::transform(prelim_results_start, prelim_results_end, prelim_results_indexes_start, prelim_results_indexes_start, index_to_keep_paired(copy_offset));
    thrust::sort(prelim_results_indexes_start, prelim_results_indexes_end, thrust::greater<int>());
    thrust::device_ptr<int> keepers_end = thrust::min_element(prelim_results_indexes_start, prelim_results_indexes_end);

    cudaSafeCall( hipMemcpy(prelim_results_indexes + keepers_end_pos, d_prelim_results_indexes, (keepers_end - prelim_results_indexes_start) * sizeof(int), hipMemcpyDeviceToHost) );
    cudaSafeCall( hipGetLastError() );

    keepers_end_pos += (keepers_end - prelim_results_indexes_start);

  }

  return keepers_end_pos;

}

void RunPairedFindBindingSitesKeepScores_cleanup(char *d_reference_sequence, unsigned int *d_rvd_pairs, double *d_scoring_matrix, unsigned char *d_prelim_results, int *d_prelim_results_indexes, unsigned char *prelim_results, int *prelim_results_indexes) {
  cudaSafeCall( hipFree(d_prelim_results) );
  cudaSafeCall( hipFree(d_prelim_results_indexes) );
  cudaSafeCall( hipHostFree(prelim_results) );
  cudaSafeCall( hipHostFree(prelim_results_indexes) );
  cudaSafeCall( hipFree(d_reference_sequence) );
  cudaSafeCall( hipFree(d_rvd_pairs) );
  cudaSafeCall( hipFree(d_scoring_matrix) );
}

void RunFindBindingSitesKeepScores_init(unsigned int **d_rvd_seq_p, double **d_scoring_matrix_p, size_t *sm_pitch_p, unsigned char **d_prelim_results_p, int **d_prelim_results_indexes_p, char **d_reference_sequence_p, unsigned char **prelim_results_p, int **prelim_results_indexes_p, unsigned long *reference_window_size_p, int *score_block_x_p, int *score_block_y_p, unsigned int *rvd_seq, double **scoring_matrix, unsigned int rvd_length, unsigned int scoring_matrix_length) {
  
  unsigned int *d_rvd_seq;
  double *d_scoring_matrix;
  size_t sm_pitch;
  unsigned char *d_prelim_results;
  int *d_prelim_results_indexes;
  unsigned char *prelim_results;
  int *prelim_results_indexes;
  char *d_reference_sequence;
  // must be divisible by 32, pref power of 2
  unsigned long reference_window_size = 134217728;// 2^27
  int score_block_x;
  int score_block_y;
  
  cudaSafeCall( hipMalloc(&d_rvd_seq, 2 * PADDED_RVD_WIDTH * sizeof(unsigned int)));
  cudaSafeCall( hipMemcpy(d_rvd_seq, rvd_seq, rvd_length * sizeof(unsigned int), hipMemcpyHostToDevice) );
  
  cudaSafeCall( hipMallocPitch(&d_scoring_matrix, &sm_pitch, 5 * sizeof(double), scoring_matrix_length * sizeof(double)) );
  
  for (unsigned int i = 0; i < scoring_matrix_length; i++) {
    cudaSafeCall( hipMemcpy(ScoringMatrixRow(d_scoring_matrix, sm_pitch, i), scoring_matrix[i], sizeof(double) * 5, hipMemcpyHostToDevice) );
  }
  
  cudaSafeCall( hipMalloc(&d_reference_sequence, reference_window_size * sizeof(char)) );
  cudaSafeCall( hipMalloc(&d_prelim_results, reference_window_size * sizeof(unsigned char)) );
  cudaSafeCall( hipMalloc(&d_prelim_results_indexes, reference_window_size * sizeof(int)) );
  
  cudaSafeCall( hipHostMalloc(&prelim_results, 400000000 * sizeof(unsigned char)) );
  cudaSafeCall( hipHostMalloc(&prelim_results_indexes, 400000000 * sizeof(int)) );

  int score_blocks_needed = (reference_window_size + SCORE_THREADS_PER_BLOCK - 1) / SCORE_THREADS_PER_BLOCK;

  score_block_x = (score_blocks_needed >= MAX_BLOCKS_PER_GRID ? MAX_BLOCKS_PER_GRID : score_blocks_needed);
  score_block_y = (score_blocks_needed + (MAX_BLOCKS_PER_GRID - 1)) / MAX_BLOCKS_PER_GRID;
  
  *d_rvd_seq_p = d_rvd_seq;
  *d_scoring_matrix_p = d_scoring_matrix;
  *sm_pitch_p = sm_pitch;
  *d_prelim_results_p = d_prelim_results;
  *d_prelim_results_indexes_p = d_prelim_results_indexes;
  *prelim_results_p = prelim_results;
  *prelim_results_indexes_p = prelim_results_indexes;
  *d_reference_sequence_p = d_reference_sequence;
  *reference_window_size_p = reference_window_size;
  *score_block_x_p = score_block_x;
  *score_block_y_p = score_block_y;
  
}

int RunFindBindingSitesKeepScores(char *d_reference_sequence, unsigned int *d_rvd_seq, double *d_scoring_matrix, size_t sm_pitch, unsigned char *d_prelim_results, int *d_prelim_results_indexes, unsigned char *prelim_results, int *prelim_results_indexes, unsigned long reference_window_size, int score_block_x, int score_block_y, unsigned int rvd_length, char *ref_seq, unsigned long ref_seq_len, double cutoff, int c_upstream) {
  
  dim3 score_threadsPerBlock(32, 14);
  dim3 score_blocksPerGrid(score_block_x, score_block_y);
  
  thrust::device_ptr<int> prelim_results_indexes_start(d_prelim_results_indexes);
  thrust::device_ptr<int> prelim_results_indexes_end(d_prelim_results_indexes + reference_window_size);
  thrust::device_ptr<unsigned char> prelim_results_start(d_prelim_results);
  thrust::device_ptr<unsigned char> prelim_results_end(d_prelim_results + reference_window_size);

  int keepers_end_pos = 0;

  int usable_tile_size = reference_window_size - (((rvd_length - 1) + 31) / 32 ) * 32;

  int iterations_needed = (ref_seq_len + (usable_tile_size - 1)) / usable_tile_size;
  
  memset(prelim_results, '\0', 400000000 * sizeof(unsigned char));
  
  for (int i = 0; i < iterations_needed; i++) {

    int copy_offset = usable_tile_size * i;

    int copy_num;

    if (ref_seq_len - copy_offset <= reference_window_size)
      copy_num = ref_seq_len - copy_offset;
    else
      copy_num = reference_window_size;

    cudaSafeCall( hipMemset(d_reference_sequence, 'X', reference_window_size * sizeof(unsigned char)) );
    cudaSafeCall( hipMemset(d_reference_sequence + reference_window_size - 1, '\0', 1 * sizeof(unsigned char)) );
    cudaSafeCall( hipMemcpy(d_reference_sequence, ref_seq + copy_offset, copy_num * sizeof(char), hipMemcpyHostToDevice) );

    cudaSafeCall( hipMemset(d_prelim_results, '\0', reference_window_size * sizeof(unsigned char)) );
    cudaSafeCall( hipMemset(d_prelim_results_indexes, '\0', reference_window_size * sizeof(unsigned int)) );
    
    ScoreBindingSites <<<score_blocksPerGrid, score_threadsPerBlock>>>(d_reference_sequence, reference_window_size, d_rvd_seq, rvd_length, cutoff, c_upstream, 0, d_scoring_matrix, sm_pitch, d_prelim_results);
    cudaSafeCall( hipGetLastError() );

    cudaSafeCall( hipMemcpy(prelim_results + copy_offset, d_prelim_results, copy_num * sizeof(unsigned char), hipMemcpyDeviceToHost) );
    cudaSafeCall( hipGetLastError() );

    thrust::sequence(prelim_results_indexes_start, prelim_results_indexes_end);
    thrust::transform(prelim_results_start, prelim_results_end, prelim_results_indexes_start, prelim_results_indexes_start, index_to_keep(copy_offset));
    thrust::sort(prelim_results_indexes_start, prelim_results_indexes_end, thrust::greater<int>());
    thrust::device_ptr<int> keepers_end = thrust::min_element(prelim_results_indexes_start, prelim_results_indexes_end);

    cudaSafeCall( hipMemcpy(prelim_results_indexes + keepers_end_pos, d_prelim_results_indexes, (keepers_end - prelim_results_indexes_start) * sizeof(int), hipMemcpyDeviceToHost) );
    cudaSafeCall( hipGetLastError() );

    keepers_end_pos += (keepers_end - prelim_results_indexes_start);

  }

  return keepers_end_pos;

}

void RunFindBindingSitesKeepScores_cleanup(char *d_reference_sequence, unsigned int *d_rvd_seq, double *d_scoring_matrix, unsigned char *d_prelim_results, int *d_prelim_results_indexes, unsigned char *prelim_results, int *prelim_results_indexes) {
  cudaSafeCall( hipFree(d_prelim_results) );
  cudaSafeCall( hipFree(d_prelim_results_indexes) );
  cudaSafeCall( hipHostFree(prelim_results) );
  cudaSafeCall( hipHostFree(prelim_results_indexes) );
  cudaSafeCall( hipFree(d_reference_sequence) );
  cudaSafeCall( hipFree(d_rvd_seq) );
  cudaSafeCall( hipFree(d_scoring_matrix) );
}


