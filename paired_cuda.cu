#include "hip/hip_runtime.h"
#undef _GLIBCXX_USE_INT128
#undef _GLIBCXX_ATOMIC_BUILTINS

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include "paired_cuda.h"

// Sequence handling
#include <zlib.h>
#include "kseq.h"
KSEQ_INIT(gzFile, gzread)

#define MAX_THREADS_PER_BLOCK 1024
#define MAX_BLOCKS_PER_GRID 65535

#define cudaSafeCall(call){   \
  hipError_t err = call;       \
  if(hipSuccess != err){     \
    fprintf(stderr, "%s(%i) : %s.\n", __FILE__, __LINE__, hipGetErrorString(err));   \
    exit(EXIT_FAILURE);       \
}}

__device__ double ScoringMatrixVal(double *scoring_matrix, size_t pitch, unsigned int row, unsigned int column);
double *ScoringMatrixRow(double *scoring_matrix, size_t pitch, unsigned int row);

//template< unsigned int STRAND, unsigned int RVD_NUM >
__global__ void ScoreBindingSites(char *input_sequence, unsigned long is_length, unsigned int *rvd_sequence, unsigned int rs_len, double cutoff, unsigned int strand, unsigned int rvd_num, double *scoring_matrix, size_t sm_pitch, unsigned char *results) {
   
  int block_seq_index = MAX_THREADS_PER_BLOCK * (blockIdx.y * gridDim.x + blockIdx.x);
  int seq_index = block_seq_index + (blockDim.x * threadIdx.y) + threadIdx.x;
  
  if (seq_index < 1 || seq_index >= is_length || seq_index + rs_len >= is_length - 1) return;
  
  if (input_sequence[seq_index - 1] == 'T' || input_sequence[seq_index - 1] == 't') {
    
    double thread_result = 0;
    
    for (int i = 0; i < rs_len; i++) {
      
      int rvd_index = i;
      
      int sm_col = 0;
      
      char base = input_sequence[seq_index + i];
      
      if (base == 'A' || base == 'a')    
        sm_col = 0;
      if (base == 'C' || base == 'c')
        sm_col = 1;
      if (base == 'G' || base == 'g')
        sm_col = 2;
      if (base == 'T' || base == 't')
        sm_col = 3;
      
      thread_result += ScoringMatrixVal(scoring_matrix, sm_pitch, rvd_sequence[rvd_index], sm_col);
      
    }
    
    results[seq_index] |= (thread_result < cutoff ? 1UL : 0UL) << (2 * rvd_num + 0);
    
  } 
  
  if (input_sequence[seq_index + rs_len] == 'A' || input_sequence[seq_index + rs_len] == 'a') {
    
    double thread_result = 0;
    
    for (int i = 0; i < rs_len; i++) {
      
      int rvd_index = rs_len - i - 1;
      
      int sm_col = 0;
      
      char base = input_sequence[seq_index + i];
      
      if (base == 'A' || base == 'a')    
        sm_col = 3;
      if (base == 'C' || base == 'c')
        sm_col = 2;
      if (base == 'G' || base == 'g')
        sm_col = 1;
      if (base == 'T' || base == 't')
        sm_col = 0;
      
      thread_result += ScoringMatrixVal(scoring_matrix, sm_pitch, rvd_sequence[rvd_index], sm_col);
      
    }
    
    results[seq_index] |= (thread_result < cutoff ? 1UL : 0UL) << (2 * rvd_num + 1);
    
  }
  
}

__global__ void TallyResults(unsigned char *prelim_results, unsigned int pr_length, unsigned int rs_len, unsigned int u_shift, unsigned int d_shift, unsigned int spacer_range_start, unsigned int spacer_range_end, unsigned int *second_results) {
    
  int thread_result = 0;
  
  int block_seq_index = MAX_THREADS_PER_BLOCK * (blockIdx.y * gridDim.x + blockIdx.x);
  int seq_index = block_seq_index + (blockDim.x * threadIdx.y) + threadIdx.x;
  
  if (seq_index < 0 || seq_index >= pr_length) return;
  
  for (int i = spacer_range_start; i <= spacer_range_end; i++) {
    
    if (seq_index + rs_len + i >= pr_length) continue;
    
    thread_result += ((prelim_results[seq_index] & (1UL << u_shift)) && 
                      (prelim_results[seq_index + rs_len + i] & (1UL << d_shift)));
  }
  
  second_results[seq_index] = thread_result;
  
}

__device__ double ScoringMatrixVal(double *scoring_matrix, size_t pitch, unsigned int row, unsigned int column) {
  
  return *((double*)((char*) scoring_matrix + row * pitch) + column);
  
}

double *ScoringMatrixRow(double *scoring_matrix, size_t pitch, unsigned int row) {
  return (double*)((char*) scoring_matrix + row * pitch);
}

void printDeviceMatrix(double *matrix, int width, int length) {
  for (int y = 0; y < length; y++) {
    double *row = ScoringMatrixRow(matrix, width, y);
    printf("[%.2f, %.2f, %.2f, %.2f]\n",
           row[0],
           row[1],
           row[2],
           row[3]);
  }
}

void printHostMatrix(double **array, int width, int length) {
  for (int y = 0; y < length; y++) {
    printf("[%.2f, %.2f, %.2f, %.2f, %.2f]\n",
           array[y][0],
           array[y][1],
           array[y][2],
           array[y][3],
           array[y][4]);
  }
}

void printRvdArray(unsigned int *array) {
  for (int y = 0; y < 32; y++) {
    printf("%du ", array[y]);
  }
  printf("\n");
}

void RunCountBindingSites(char *seq_filename, unsigned int *spacer_sizes, unsigned int **rvd_sequences, unsigned int *rvd_sequence_lengths, double *cutoffs, double **scoring_matrix, unsigned int scoring_matrix_length, unsigned int **results) {
  
  unsigned int *d_rvd_sequence;
  unsigned int *d_rvd_sequence2;
  double *d_scoring_matrix;
  size_t sm_pitch;
  hipEvent_t start, stop;
  float elapsed;
  
  cudaSafeCall( hipMalloc(&d_rvd_sequence, 32 * sizeof(unsigned int)));
  cudaSafeCall( hipMemcpy(d_rvd_sequence, rvd_sequences[0], 32 * sizeof(unsigned int), hipMemcpyHostToDevice) );

  cudaSafeCall( hipMalloc(&d_rvd_sequence2, 32 * sizeof(unsigned int)));
  cudaSafeCall( hipMemcpy(d_rvd_sequence2, rvd_sequences[1], 32 * sizeof(unsigned int), hipMemcpyHostToDevice) );
    
  cudaSafeCall( hipMallocPitch(&d_scoring_matrix, &sm_pitch, 4 * sizeof(double), scoring_matrix_length * sizeof(double)) );
  
  for (unsigned int i = 0; i < scoring_matrix_length; i++) {
    cudaSafeCall( hipMemcpy(ScoringMatrixRow(d_scoring_matrix, sm_pitch, i), scoring_matrix[i], sizeof(double) * 4, hipMemcpyHostToDevice) );
  }
  
  dim3 threadsPerBlock(32, 32);
  
  gzFile seqfile = gzopen(seq_filename, "r");

  kseq_t *seq = kseq_init(seqfile);
  int result;

  while ((result = kseq_read(seq)) >= 0) {
    
    unsigned char *d_prelim_results;
    unsigned int *d_second_results;
    char *d_reference_sequence;
    unsigned int h_results[4];
    
    char *reference_sequence = seq->seq.s;
    
    for (int i = seq->seq.l; i < seq->seq.m; i++) {
      reference_sequence[i] = 'X';
    }
    
    reference_sequence[seq->seq.m - 1] = '\0';
    
    unsigned long reference_sequence_length = seq->seq.m;
    
    cudaSafeCall( hipMalloc(&d_reference_sequence, reference_sequence_length * sizeof(char)) );
    cudaSafeCall( hipMemcpy(d_reference_sequence, reference_sequence, reference_sequence_length * sizeof(char), hipMemcpyHostToDevice) );
  
    cudaSafeCall( hipMalloc(&d_prelim_results, reference_sequence_length * sizeof(unsigned char)) );
    cudaSafeCall( hipMemset(d_prelim_results, '\0', reference_sequence_length * sizeof(unsigned char)) );
    
    cudaSafeCall( hipMalloc(&d_second_results, reference_sequence_length * sizeof(unsigned int)) );
    cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
    
    thrust::device_ptr<unsigned int> second_results_start(d_second_results);
    thrust::device_ptr<unsigned int> second_results_end(d_second_results + reference_sequence_length);

    int blocks_needed = reference_sequence_length / 1024;
    int block_y = (blocks_needed + (MAX_BLOCKS_PER_GRID - 1)) / MAX_BLOCKS_PER_GRID;
    dim3 blocksPerGrid(MAX_BLOCKS_PER_GRID, block_y);
    
    cudaSafeCall( hipEventCreate(&start) );
    cudaSafeCall( hipEventCreate(&stop) );
    
    cudaSafeCall( hipEventRecord(start, 0) );
    
    ScoreBindingSites <<<blocksPerGrid, threadsPerBlock>>>(d_reference_sequence, reference_sequence_length, d_rvd_sequence, rvd_sequence_lengths[0], cutoffs[0], 0, 0, d_scoring_matrix, sm_pitch, d_prelim_results);
    cudaSafeCall( hipGetLastError() );

    ScoreBindingSites <<<blocksPerGrid, threadsPerBlock>>>(d_reference_sequence, reference_sequence_length, d_rvd_sequence2, rvd_sequence_lengths[1], cutoffs[1], 0, 1, d_scoring_matrix, sm_pitch, d_prelim_results);
    cudaSafeCall( hipGetLastError() );
    
    cudaSafeCall( hipDeviceSynchronize() );
    
    cudaSafeCall( hipEventRecord(stop, 0) );
    cudaSafeCall( hipEventSynchronize(stop) );
    cudaSafeCall( hipEventElapsedTime(&elapsed, start, stop) );
    
    printf("%.2f ms to score binding sites\n", elapsed);
    
    cudaSafeCall( hipEventDestroy(stop) );
    cudaSafeCall( hipEventDestroy(start) );
    
    
    cudaSafeCall( hipEventCreate(&start) );
    cudaSafeCall( hipEventCreate(&stop) );
    
    cudaSafeCall( hipEventRecord(start, 0) );
    
    TallyResults<<<blocksPerGrid, threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_sequence_lengths[0], 0, 1, spacer_sizes[0], spacer_sizes[1], d_second_results);
    cudaSafeCall( hipGetLastError() );
    
    h_results[0] = thrust::reduce(second_results_start, second_results_end);
    cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
    
    TallyResults<<<blocksPerGrid, threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_sequence_lengths[0], 0, 3, spacer_sizes[0], spacer_sizes[1], d_second_results);
    cudaSafeCall( hipGetLastError() );
    
    h_results[1] = thrust::reduce(second_results_start, second_results_end);
    cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
    
    TallyResults<<<blocksPerGrid, threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_sequence_lengths[1], 2, 1, spacer_sizes[0], spacer_sizes[1], d_second_results);
    cudaSafeCall( hipGetLastError() );
    
    h_results[2] = thrust::reduce(second_results_start, second_results_end);
    cudaSafeCall( hipMemset(d_second_results, '\0', reference_sequence_length * sizeof(unsigned int)) );
    
    TallyResults<<<blocksPerGrid, threadsPerBlock>>>(d_prelim_results, reference_sequence_length, rvd_sequence_lengths[1], 2, 3, spacer_sizes[0], spacer_sizes[1], d_second_results);
    cudaSafeCall( hipGetLastError() );
    
    h_results[3] = thrust::reduce(second_results_start, second_results_end);
    
    cudaSafeCall( hipDeviceSynchronize() );
    
    cudaSafeCall( hipEventRecord(stop, 0) );
    cudaSafeCall( hipEventSynchronize(stop) );
    
    cudaSafeCall( hipEventElapsedTime(&elapsed, start, stop) );
    
    printf("%.2f ms to tally results\n", elapsed);
    
    cudaSafeCall( hipEventDestroy(stop) );
    cudaSafeCall( hipEventDestroy(start) );
  
    results[0][0] += h_results[0];
    results[0][1] += h_results[1];
    results[1][0] += h_results[2];
    results[1][1] += h_results[3];
    
    printf("%d %d %d %d\n", h_results[0], h_results[1], h_results[2], h_results[3]);
    
    cudaSafeCall( hipFree(d_prelim_results) );
    cudaSafeCall( hipFree(d_second_results) );
    cudaSafeCall( hipFree(d_reference_sequence) );
    
  }

  kseq_destroy(seq);
  gzclose(seqfile);
  
  cudaSafeCall( hipFree(d_rvd_sequence) );
  cudaSafeCall( hipFree(d_rvd_sequence2) );
  cudaSafeCall( hipFree(d_scoring_matrix) );
  
}
